#include "hip/hip_runtime.h"
#include <stdio.h>
#define _size 512
__global__ void mul(int *a, int *b, int *c)
{
	c[threadIdx.x + blockIdx.x*blockDim.x] = a[threadIdx.x + blockIdx.x*blockDim.x]*b[threadIdx.x + blockIdx.x*blockDim.x];
}

int main()
{
	int SIZE = _size*sizeof(int);
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	a = (int *)malloc(SIZE);
	b = (int *)malloc(SIZE);
	c = (int *)malloc(SIZE);
	hipMalloc((void **)&d_a,SIZE);
	hipMalloc((void **)&d_b,SIZE);
	hipMalloc((void **)&d_c,SIZE);
	printf("Enter value of A :\n");
	for(int i=0;i<_size;i++)
	{
			a[i]=i*2;
			//printf("a[%d]\t",i);
			//scanf("%d",&a[i]);
	}
	for(int i=0;i<_size;i++)
	{
			b[i]=i*2+12;
			//printf("b[%d]\t",i);
			//scanf("%d",&b[i]);
	}
		for(int i=0;i<_size;i++)
	{

			printf("a[%d]: %d\tb[%d]: %d\t",i,a[i],i,b[i]);
	}
		printf("\n");
	hipMemcpy(d_a,a,SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,SIZE,hipMemcpyHostToDevice);
	mul<<<2,256>>>(d_a,d_b,d_c);
	hipDeviceSynchronize();
	hipMemcpy(c,d_c,SIZE,hipMemcpyDeviceToHost);
	printf("\n");
	for(int i=0;i<_size;i++)
	{

			printf("c[%d]: %d\t",i,c[i]);
	}
	return 0;
}
