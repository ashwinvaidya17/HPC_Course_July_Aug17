#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define _size 3	//Define size of matrix to be 3 by 3;
//__shared__ int result[_size*_size];
__global__ void multiply(int *result, int *A, int *B)
{
	/* OLD logic
	We have a 3 by 3 grid and each block has 3 threads.
	So rows = block x id, cols = block y id
	So Indices will be C[block X id][block Y id] = A[block X id][threads 0, 1, 2] * B[threads 0, 1, 2][block y id]
	*/
	//__shared__ int result[_size*_size] ;
	/*result[blockIdx.x*blockDim.x +blockIdx.y] += A[blockIdx.x*blockDim.x + threadIdx.x]*B[blockDim.x*threadIdx.x+blockIdx.y];
	printf("C[%d] = A[%d]*B[%d] = %d*%d\n",blockIdx.x*blockDim.x +blockIdx.y, blockIdx.x*blockDim.x + threadIdx.x, blockDim.x*threadIdx.x+blockIdx.y,
		A[blockIdx.x*blockDim.x + threadIdx.x],B[blockDim.x*threadIdx.x+blockIdx.y]);
	Res[blockIdx.x*blockDim.x +blockIdx.y]= result[blockIdx.x*blockDim.x +blockIdx.y];*/
	
	/* NEW logic
	I have 3 blocks and 3 threads. Each thread calculates entry for each position compared to the old one having each thread multiplying one value.
	So indices will be result[block x id][thread id] = A[block x id][i]* B[i][thread x id] 
	*/

	for(int i=0; i<_size;i++)
	{
		result[blockIdx.x*blockDim.x +threadIdx.x] += A[blockIdx.x*blockDim.x+i]*B[blockDim.x*i+threadIdx.x];
	}
}

int main(int argc, char const *argv[])
{
	int *d_c,*d_a,*d_b;
	int size = _size*sizeof(int)*_size;
	int *a = reinterpret_cast<int*>(malloc(size));
	int *b = reinterpret_cast<int*>(malloc(size));
	//*result = reinterpret_cast<int*>(malloc(size));
	int *c = reinterpret_cast<int*>(malloc(size));
	//Getting Matrix A
	printf("Enter matrix A\n");
	for (int i = 0; i < _size; ++i)
	{
		for (int j = 0; j < _size; ++j)
		{
			scanf("%d",&*(a+i*_size+j) );
		}
	}
	//Printing matrix A
	printf("A is: \n");
	for (int i = 0; i < _size; ++i)
	{
		for (int j = 0; j < _size; ++j)
		{
			printf("%d ",*(a+i*_size+j) );
		}
		printf("\n");
	}
	//Gtting matrix B
	printf("Enter matrix B\n");
	for (int i = 0; i < _size; ++i)
	{
		for (int j = 0; j < _size; ++j)
		{
			scanf("%d",&*(b+i*_size+j));
		}
	}
	//Printing matrix B
	printf("B is: \n");
	for (int i = 0; i < _size; ++i)
	{
		for (int j = 0; j < _size; ++j)
		{
			printf("%d ",*(b+i*_size+j) );
		}
		printf("\n");
	}
	memset(c,0,size);
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);
	multiply<<<_size,_size>>>(d_c,d_a,d_b);
	hipDeviceSynchronize();
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("C is: \n");
	for (int i = 0; i < _size; ++i)
	{
		for (int j = 0; j < _size; ++j)
		{
			printf("%d ",*(c+i*_size+j) );
		}
		printf("\n");
	}
	return 0;
}